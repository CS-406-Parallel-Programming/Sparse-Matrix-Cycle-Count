//#include "scale.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <math.h> /* fabs */
#include <string.h>
#include <stdlib.h>
#include <sstream>
#include <unordered_map>

using namespace std;
#define THREADS_PER_BLOCK 256
#define STREAM_COUNT 4
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

int flag;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}


void printArray(int *arr, int nov){
    for (int j=0;j<nov;j++)
    {
        cout << j << " " << arr[j] << endl;
    }
}

__global__ void kernel3(int* adj, int* xadj, int* output, int nov){
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if(index < nov){
      //int *marked = new int[n];
      //memset(marked, -1, n * sizeof(int)); // bu belki silinebilir
      int localcount = 0;
      // int round = 0;

      // 0-->
      int s0 = xadj[index];
      int e0 = xadj[index+1];

      for(int i=s0; i < e0; i++){
        // 0 --> 1

        int neighbour_1 = adj[i];
        int s1  = xadj[neighbour_1];
        int e1  = xadj[neighbour_1+1];

        for(int j=s1;j < e1; j++){
          // 0 --> 1 --> 2

          int neighbour_2 =  adj[j];
          if (neighbour_2 == index) continue;
          int s2  = xadj[neighbour_2];
          int e2  = xadj[neighbour_2+1];

          for(int k=s2; k < e2; k++){

            // 0 --> 1 --> 2 --> 3
            int neighbour_3 =  adj[k];
            if (neighbour_3 == index){
              localcount+=1;
              break;
            }
          }
        }
      }
      output[index] = localcount;
    }
}

__global__ void kernel4(int* adj, int* xadj, int* output, int nov){
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if(index < nov){
      //int *marked = new int[n];
      //memset(marked, -1, n * sizeof(int)); // bu belki silinebilir
      int localcount = 0;
      // int round = 0;

      // 0-->
      int s0 = xadj[index];
      int e0 = xadj[index+1];

      for(int i=s0; i < e0; i++){
        // 0 --> 1

        int neighbour_1 = adj[i];
        int s1  = xadj[neighbour_1];
        int e1  = xadj[neighbour_1+1];

        for(int j=s1;j < e1; j++){
          // 0 --> 1 --> 2

          int neighbour_2 =  adj[j];
          //eliminate 0 == 2
          if (neighbour_2 == index) continue;
          int s2  = xadj[neighbour_2];
          int e2  = xadj[neighbour_2+1];

          for(int k=s2; k < e2; k++){

            // 0 --> 1 --> 2 --> 3

            int neighbour_3 =  adj[k];
            //eliminate 3 == 0
            if (neighbour_3 == index) continue;
            // eliminate 3 ==1
            if (neighbour_3 == neighbour_1) continue;
            int s3  = xadj[neighbour_3];
            int e3  = xadj[neighbour_3+1];


            for(int n=s3; n < e3; n++){
              //0 -->1 -->2 -->3 -->4


              int neighbour_4 =  adj[n];
              if (neighbour_4 == index){
                localcount+=1;
                break;
              }
            }
          }
        }
      }
      output[index] = localcount;
    }
}
__global__ void kernel5(int* adj, int* xadj, int* output, int nov){
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if(index < nov){
      //int *marked = new int[n];
      //memset(marked, -1, n * sizeof(int)); // bu belki silinebilir
      int localcount = 0;
      // int round = 0;

      // 0-->
      int s0 = xadj[index];
      int e0 = xadj[index+1];

      for(int i=s0; i < e0; i++){
        // 0 --> 1

        int neighbour_1 = adj[i];
        int s1  = xadj[neighbour_1];
        int e1  = xadj[neighbour_1+1];

        for(int j=s1;j < e1; j++){
          // 0 --> 1 --> 2

          int neighbour_2 =  adj[j];
          //eliminate 0 == 2
          if (neighbour_2 == index) continue;
          int s2  = xadj[neighbour_2];
          int e2  = xadj[neighbour_2+1];

          for(int k=s2; k < e2; k++){

            // 0 --> 1 --> 2 --> 3

            int neighbour_3 =  adj[k];
            //eliminate 3 == 0
            if (neighbour_3 == index) continue;
            // eliminate 3 ==1
            if (neighbour_3 == neighbour_1) continue;
            int s3  = xadj[neighbour_3];
            int e3  = xadj[neighbour_3+1];


            for(int n=s3; n < e3; n++){
              //0 -->1 -->2 -->3 -->4


              int neighbour_4 =  adj[n];
              //eliminate 4 == 0
              if (neighbour_4 == index) continue;
              // eliminate 4 ==1
              if (neighbour_4 == neighbour_1) continue;
              // eliminate 4 ==2
              if (neighbour_4 == neighbour_2) continue;

              int s4  = xadj[neighbour_4];
              int e4  = xadj[neighbour_4+1];


              for(int o=s4; o < e4; o++){

                //0 -->1 -->2 -->3 -->4--> 5

                int neighbour_5 =  adj[o];
                if (neighbour_5 == index){
                  localcount+=1;
                  break;
                }
              }
            }
          }
        }
      }
      output[index] = localcount;
    }
}

void wrapper(int *xadj, int *adj, int n,  int nov, int nnz){

  // int X = nov;
  // int Y = maxSize;
  // int Z = maxSize;
  //
  // dim3 threadsPerBlock(8, 8, 8);
  // dim3 numBlocks(X/threadsPerBlock.x,  /* for instance 512/8 = 64
  //                Y /threadsPerBlock.y,
  //                Z/threadsPerBlock.z);
  //





  hipSetDevice(0);
  int *adj_d;
  int *xadj_d;
  int *output_d;
  int *output_h = new int[nov];
  int numBlock = (nov + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  hipEvent_t start, stop;
  float elapsedTime;

/*
  int novForThread = (nov+STREAM_COUNT-1)/STREAM_COUNT;
  int novStart = novForThread * threadId;
  int novEnd   = novForThread * (threadId+1);
  if (novEnd> nov) novEnd = nov;
  int numBlock = (novEnd-novStart + THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;
*/



  gpuErrchk(hipMalloc((void**)&adj_d, (nnz) * sizeof(int)));
  gpuErrchk(hipMalloc((void**)&xadj_d, (nov + 1) * sizeof(int)));

  gpuErrchk(hipMalloc((void**)&output_d, (nov) * sizeof(int)));

  //gpuErrchk(cudaMallocHost((void **)&output_h, (nov) * sizeof(int)));

  gpuErrchk(hipMemcpy(adj_d, adj, (nnz) * sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(xadj_d, xadj, (nov + 1) * sizeof(int), hipMemcpyHostToDevice));

  hipEventCreate(&start);
  hipEventRecord(start, 0);


  



  if (n==3)       kernel3<<<numBlock, THREADS_PER_BLOCK>>>(adj_d, xadj_d, output_d, nov);
  else if (n==4)  kernel4<<<numBlock, THREADS_PER_BLOCK>>>(adj_d, xadj_d, output_d, nov);
  else if (n==5)  kernel5<<<numBlock, THREADS_PER_BLOCK>>>(adj_d, xadj_d, output_d, nov);

  //combination<<<numBlocks, threadsPerBlock>>>(adj_d, xadj_d, output_d, n, nov);

  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(output_h, output_d, (nov) * sizeof(int), hipMemcpyDeviceToHost));

  hipEventCreate(&stop);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&elapsedTime, start, stop);

  if(flag == 0)	printArray(output_h,nov);
  hipFree(adj_d);
  hipFree(xadj_d);
  if(flag == 1)	printf("GPU scale took: %f s\n", elapsedTime/1000);
}


void  read_mtxbin(string fname, int k){
  //cout << "fname: " << fname << endl;
  ifstream infile(fname);
  int a, b;
  int nnv = 0;
  unordered_map<int, vector<int> > hashmap;

  int  maxElement  = -1;

  while (infile >> a >> b)
  {
      nnv+=2;
      hashmap[a].push_back(b);
      hashmap[b].push_back(a);

      if(b > maxElement){
        maxElement = b;
      }
  }
  //cout << end1-start1 << " -- ILK OKUMA SU (s).\n";

  int nov = maxElement +1;
  //cout <<"nov " << nov << endl;
  //cout <<"nnv " << nnv << endl;
  int * adj  = new int[nnv];
  int * xadj = new int[nov+1];
  xadj[0]=0;

  int j = 0;
  int maxSize = -1;

  for(int i=0; i < nov ; i++ ){
    auto current = hashmap.find(i);
    if (current == hashmap.end()){
        xadj[i+1] = xadj[i];
    }
    else{
        int size = current->second.size();
        maxSize = max(size,maxSize);

        xadj[i+1] = xadj[i] + size;
        for(auto val : current->second) {
            adj[j] = val;
            j++;
        }
    }
  }
  // cout << "maxSize: "<<maxSize<<endl;
  // cout << end-start << " -- OKUMA SURE (s).\n";
  wrapper(xadj,adj,k,nov,nnv);
  //cout<<"CYCLES: --> "<<countCycles_sparse(xadj, adj,k,nov)<<endl;

  /*double end2 = omp_get_wtime();
  cout << end2-start << " -- TOTAL SURE (s).\n";*/
}

int main(int argc, char *argv[]){
    char* fname = argv[1];
    int k = atoi(argv[2]);
    flag = atoi(argv[3]);
    read_mtxbin(fname,k);
    return 0;
}
